#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 10000
#define BLOCK_SIZE 256

// CUDA Kernel function to add the elements of two arrays
__global__ void add(int *a, int *b, int *c) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  c[index] = a[index] + b[index];
}

void fill_array(int *data, int size){
  for(int i=0;i<size;i++)
    data[i] = i;
}

int main(void) {
  int *a, *b, *c; // host copies of a, b, c
  int *d_a, *d_b, *d_c; // device copies of a, b, c
  int size = N * sizeof(int);

  // Allocate space for device copies of a, b, c
  hipMalloc((void **)&d_a, size);
  hipMalloc((void **)&d_b, size);
  hipMalloc((void **)&d_c, size);

  // Setup input values
  a = (int*)malloc(size); fill_array(a, N);
  b = (int*)malloc(size); fill_array(b, N);
  c = (int*)malloc(size);

  // Copy inputs to device
  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

  // Launch add() kernel on GPU with N blocks
  add<<<N/BLOCK_SIZE, BLOCK_SIZE>>>(d_a, d_b, d_c);

  // Copy result back to host
  hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

  // Cleanup
  free(a); free(b); free(c);
  hipFree(d_a); hipFree(d_b); hipFree(d_c);

  return 0;
}
